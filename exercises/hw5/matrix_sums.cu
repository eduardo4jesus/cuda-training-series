
#include <hip/hip_runtime.h>
#include <stdio.h>

// error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


// In case of debugging
// const size_t DSIZE = 8;
// const int block_size = 4;  // CUDA maximum is 1024
const size_t DSIZE = 16384;      // matrix side dimension
const int block_size = 256;  // CUDA maximum is 1024
// matrix row-sum kernel
__global__ void row_sums(const float *A, float *sums, size_t ds) {

  /** 1. Each block will compute a row-sweep reduction sum */
  int idrow = blockIdx.x;
  
  /** 2. Allocating and reseting the shared memory values */
  __shared__ float sdata[block_size];
  size_t tid = threadIdx.x;
  sdata[tid] = 0;

  /** 3. Block loop to populate the shared memory with the row data */
  int idcol = tid;
  if (idrow < ds) {
    while (idcol < ds) {
      sdata[tid] += A[idrow*ds + idcol];
      idcol += blockDim.x;
    }
  }

  // printf("Processing row %d.\n", idrow);
  // if (idrow == 2 && tid == 0) {
  //   printf("sdata row %d:\n", idrow);
  //   for(int i=0; i<block_size; i++)
  //     printf("%.3f ", sdata[i]);
  //   printf("\n");
  // }

  /** 4. Now, let's compute the row (block) sum by doing a row-reduction sweep */
  // int s_width = min((size_t) blockDim.x, ds);
  for (unsigned int s = blockDim.x/ 2; s > 0; s >>= 1) {
    __syncthreads();
    if (tid < s) {
      sdata[tid] += sdata[tid + s];
    }
  }

  if (tid == 0 && idrow < ds) {
    sums[idrow] = sdata[0];
  }
}

__global__ void row_sums_slow(const float *A, float *sums, size_t ds) {

  /** 1. let's define a shared memory area */
  __shared__ float sdata[block_size];
  size_t tid = threadIdx.x;

  /** 2. Each thread will iterate over the rows, summing each one at a time */
  for (size_t idrow = 0; idrow < ds; idrow++) {
    __syncthreads();  
    sdata[tid] = 0;

    /** 3. Let's do a grid loop to populate the shared memory with the row data */
    int idcol = threadIdx.x + blockDim.x * blockIdx.x;
    while (idcol < ds) {
      sdata[tid] += A[idrow * ds + idcol];
      idcol += gridDim.x * blockDim.x;
    }

    /** 4. Now, let's compute the row partial sum for each block by doing a row-reduction sweep */
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
      __syncthreads();
      if (tid < s) {
        sdata[tid] += sdata[tid + s];
      }
    }

    if (tid == 0) {
      atomicAdd(&sums[idrow], sdata[0]);
    }
  }
}

__global__ void row_sums_original(const float *A, float *sums, size_t ds) {
  int idx = threadIdx.x+blockDim.x*blockIdx.x; // create typical 1D thread index from built-in variables
  if (idx < ds){
    float sum = 0.0f;
    for (size_t i = 0; i < ds; i++)
      sum += A[idx*ds+i];         // write a for loop that will cause the thread to iterate across a row, keeeping a running sum, and write the result to sums
    sums[idx] = sum;
  }
}

// matrix column-sum kernel
__global__ void column_sums(const float *A, float *sums, size_t ds){

  int idx = threadIdx.x+blockDim.x*blockIdx.x; // create typical 1D thread index from built-in variables
  if (idx < ds){
    float sum = 0.0f;
    for (size_t i = 0; i < ds; i++)
      sum += A[idx+ds*i];         // write a for loop that will cause the thread to iterate down a column, keeeping a running sum, and write the result to sums
    sums[idx] = sum;
}}
bool validate(float *data, size_t sz){
  for (size_t i = 0; i < sz; i++)
    if (data[i] != (float)sz) {printf("results mismatch at %lu, was: %f, should be: %f\n", i, data[i], (float)sz); return false;}
    return true;
}
int main(){

  float *h_A, *h_sums, *d_A, *d_sums;
  h_A = new float[DSIZE*DSIZE];  // allocate space for data in host memory
  h_sums = new float[DSIZE]();
  for (int i = 0; i < DSIZE*DSIZE; i++)  // initialize matrix in host memory
    h_A[i] = 1.0f;
  hipMalloc(&d_A, DSIZE*DSIZE*sizeof(float));  // allocate device space for A
  hipMalloc(&d_sums, DSIZE*sizeof(float));  // allocate device space for vector d_sums
  cudaCheckErrors("hipMalloc failure"); // error checking
  // copy matrix A to device:
  hipMemcpy(d_A, h_A, DSIZE*DSIZE*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");
  //cuda processing sequence step 1 is complete
  row_sums<<<DSIZE, block_size>>>(d_A, d_sums, DSIZE);
  // row_sums<<<(DSIZE+block_size-1)/block_size, block_size>>>(d_A, d_sums, DSIZE);
  cudaCheckErrors("kernel launch failure");
  //cuda processing sequence step 2 is complete
  // copy vector sums from device to host:
  hipMemcpy(h_sums, d_sums, DSIZE*sizeof(float), hipMemcpyDeviceToHost);
  //cuda processing sequence step 3 is complete
  cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
  if (!validate(h_sums, DSIZE)) return -1; 
  printf("row sums correct!\n");
  hipMemset(d_sums, 0, DSIZE*sizeof(float));
  column_sums<<<(DSIZE+block_size-1)/block_size, block_size>>>(d_A, d_sums, DSIZE);
  cudaCheckErrors("kernel launch failure");
  //cuda processing sequence step 2 is complete
  // copy vector sums from device to host:
  hipMemcpy(h_sums, d_sums, DSIZE*sizeof(float), hipMemcpyDeviceToHost);
  //cuda processing sequence step 3 is complete
  cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
  if (!validate(h_sums, DSIZE)) return -1; 
  printf("column sums correct!\n");
  return 0;
}
  
